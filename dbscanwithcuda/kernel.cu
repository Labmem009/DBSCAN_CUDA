#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hipblas.h"
#include ""
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <windows.h>
#include <math.h>
#include <queue>

using namespace std;

struct Point {
	float		x;
	float       y;
	int			cluster;			
	int			noise;  //-1 noise

};

int eps = 2;//neighborhood radius
int min_nb = 3;
Point host_sample[500];//312
int block_num, thread_num;

float __device__ dev_euclidean_distance(const Point &src, const Point &dest) {

	float res = (src.x - dest.x) * (src.x - dest.x) + (src.y - dest.y) * (src.y - dest.y);

	return sqrt(res);
}

/*to get the total list*/
void __global__ dev_region_query(Point* sample, int num, int* neighbors, int eps, int min_nb) {

	unsigned int	tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int	line,col,pointer = tid;
	unsigned int	count;
	 
	while (pointer < num * num) {//全场唯一id
		line = pointer / num;
		col = pointer % num;
		float radius;
		if (line <= col) {
			radius = dev_euclidean_distance(sample[line], sample[col]);
			if (radius < eps) {
				neighbors[pointer] = 1;
			}
			neighbors[col * num + line] = neighbors[pointer];//对角线
		}
		pointer += blockDim.x * gridDim.x;
	}
	__syncthreads();

	pointer = tid;
	while (pointer < num) {
		count = 0;
		line = pointer * num;
		for (int i = 0; i < num; i++) {
			if (pointer != i && neighbors[line+i]) {//除了p点外邻域元素个数
					count++;
			}
		}
		if (count >= min_nb) {
			sample[pointer].noise++;
		}
		pointer += blockDim.x * gridDim.x;
	}
}

void host_algorithm_dbscan(Point* host_sample, int num) {
	/*sample*/
	Point* cuda_sample;
	hipMalloc((void**)&cuda_sample, num * sizeof(Point));
	hipMemcpy(cuda_sample, host_sample, num * sizeof(Point), hipMemcpyHostToDevice);

	/*neighbor list*/
	int *host_neighbor = new int[num*num]();
	int *dev_neighbor;
	hipMalloc((void**)&dev_neighbor, num * num * sizeof(int));
	
	dev_region_query << <block_num, thread_num >> > (cuda_sample, num, dev_neighbor, eps, min_nb);

	hipMemcpy(host_sample, cuda_sample, num * sizeof(Point), hipMemcpyDeviceToHost);
	hipMemcpy(host_neighbor, dev_neighbor, num * num * sizeof(int), hipMemcpyDeviceToHost);
	
	queue<int> expand;
	int cur_cluster = 0;

	for (int i = 0; i < num; i++) {
		if (host_sample[i].noise >= 0 && host_sample[i].cluster < 1) {
			host_sample[i].cluster = ++cur_cluster; 
			int src = i * num;
			for (int j = 0; j < num; j++) {
				if (host_neighbor[src + j]) {
					host_sample[j].cluster = cur_cluster;
					expand.push(j);
				}
			}

			while (!expand.empty()) {/*expand the cluster*/
				if (host_sample[expand.front()].noise >= 0) {
					src = expand.front() * num;
					for (int j = 0; j < num; j++) {
						if (host_neighbor[src + j] && host_sample[j].cluster < 1) {
							host_sample[j].cluster = cur_cluster;
							expand.push(j);
						}
					}
				}
				expand.pop();
			}
		}
	}
	hipFree(cuda_sample);hipFree(dev_neighbor);
}

int main(int argc, char* argv[]) {
	clock_t starts, finishs;
	double duration;
	starts = clock();
	ifstream fin("3spiral.txt");
	ofstream fout;
	fout.open("result.txt");
	int sample_num = 0;
	double a, b;
	while (fin >> a >> b) {
		host_sample[sample_num].x = a;
		host_sample[sample_num].y = b;
		host_sample[sample_num].noise = -1;
		host_sample[sample_num].cluster = -1;
		sample_num++;
	}

	cout << "------>TOTAL SAMPLE NUMB0->" << sample_num << "<-----" << endl;
	cout << "------>BL0CK=10 & THREAD=100<-------- "<< endl;
	block_num = 10;
	thread_num = 100;
	cout<<"CALCULATING BY CUDA GTX965M......\n"<<endl;
	
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);

	host_algorithm_dbscan(host_sample, sample_num);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);

	float time;
	hipEventElapsedTime(&time, start, end);
	cout<<"time: "<< time <<"ms --device\n"<<endl;
	
	finishs = clock();
	duration = (double)(finishs - starts) / CLOCKS_PER_SEC;
	cout << duration << "s --total" << endl;

	for (int i = 0; i < sample_num; i++) {
		fout <<"["<<host_sample[i].x << "," << host_sample[i].y << "] -->"<<host_sample[i].cluster<< endl;
	}

	fout.close();
	system("pause");
	return 0;
}